#include "hip/hip_runtime.h"
#include <iostream>
// in Windows 
#include <hip/hip_runtime.h>

using namespace std;

// CUDA Kernel function to add elements of two arrays
__global__ void add(int* a, int* b, int* c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    // Array size
    const int n = 10;
    int size = n * sizeof(int);

    // Host arrays
    int h_a[n], h_b[n], h_c[n];

    // Initialize arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device arrays
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    /* Define the number of threads per block and the number of blocks using dim3
       *** What is dim3? **
         The dim3 data type in CUDA is used to define the dimensions of blocks and grids.
         It allows you to specify the number of threads in each block and the number of blocks in each grid.
          You can think of dim3 as a 3D vector with x, y, and z dimensions. In most simple cases  */

    dim3 threadsPerBlock(n, 1, 1);
    dim3 blocksPerGrid(1, 1, 1);

    // Launch kernel on the GPU using dim3 configuration
    add << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Display the results
    for (int i = 0; i < n; i++) {
        cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
